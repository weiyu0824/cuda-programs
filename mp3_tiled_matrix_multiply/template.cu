
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <chrono>
#define TILE_SIZE 4

__global__ void MatMul(float *A, float *B, float *C, int M, int N, int K){
  // A = M x N, B = N x K  
  int blockRow = blockIdx.x;
  int blockCol = blockIdx.y;
  int threadRow = threadIdx.x; 
  int threadCol = threadIdx.y;
  
  int ARow = blockRow * TILE_SIZE + threadRow; 
  int BCol = blockCol * TILE_SIZE + threadCol;
  int CRow = ARow;
  int CCol = BCol;

  float val = 0; 
  for (int tileIdx = 0; tileIdx < (N+TILE_SIZE-1)/TILE_SIZE; tileIdx ++){
    // printf("Block %d.%d, Thread, %d.%d\n", blockRow, blockCol, threadRow, threadCol);
    // Cooperative load  
    __shared__ float ASub[TILE_SIZE][TILE_SIZE];
    __shared__ float BSub[TILE_SIZE][TILE_SIZE];

    int ACol = tileIdx * TILE_SIZE + threadCol; 
    int BRow = tileIdx * TILE_SIZE + threadRow;
    if (ARow < M && ACol < N){
      ASub[threadRow][threadCol] = A[ARow * N + ACol];
    }    
    if (BRow < N && BCol < N){
      BSub[threadRow][threadCol] = B[BRow * K + BCol];
    }
    __syncthreads();
    
    // Inner product  
    for (int e = 0; e < TILE_SIZE; e ++){
      val += ASub[threadRow][e] * BSub[e][threadCol];
    }

    __syncthreads();
  }
   
  if (CRow < M && CCol < K){
    C[CRow * blockDim.y + CCol] = val; 
  }
} 

void print_matrix(float* matrix, int row, int col){
    for (int i = 0; i < row; i ++) {
      for (int j = 0; j < col; j ++) {
        int idx = i * col + j;
        std::cout << matrix[idx] << " "; 
      }
      std::cout << std::endl;
    } 
}

int main(){
  // Data size
  int M = 10; 
  int N = 4;
  int K = 4; 

  size_t Mat_A_bytes = M * N * sizeof(float);
  size_t Mat_B_bytes = N * K * sizeof(float);
  size_t Mat_C_bytes = M * K * sizeof(float);

  // Init host memory
  float *host_A = (float*)malloc(Mat_A_bytes);
  float *host_B = (float*)malloc(Mat_B_bytes);
  float *host_C = (float*)malloc(Mat_C_bytes);
  
  // Init device memory
  float *device_A, *device_B, *device_C;
  hipMalloc((void**)&device_A, Mat_A_bytes);
  hipMalloc((void**)&device_B, Mat_B_bytes);
  hipMalloc((void**)&device_C, Mat_C_bytes);

  // Init Data
  for (int i = 0; i < M; i ++) {
    for (int j = 0; j < N; j ++) {
       int idx = i * N + j; 
       host_A[idx] = i;
    }
  } 

  for (int i = 0; i < N; i ++) {
    for (int j = 0; j < K; j ++) {
      int idx = i * K + j;
      host_B[idx] = i;
    }
  } 

  // Copy data to device
  hipMemcpy(device_A, host_A, Mat_A_bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, Mat_B_bytes, hipMemcpyHostToDevice);

  // Lanch kernel 
  dim3 blockDim(TILE_SIZE, TILE_SIZE);
  int numBlockRows = (int)ceil(M / (float)blockDim.x);
  int numBlockCols = (int)ceil(K / (float)blockDim.y);
  dim3 gridDim(numBlockRows, numBlockCols);

  auto start = std::chrono::high_resolution_clock::now();
  MatMul<<<gridDim, blockDim>>>(device_A, device_B, device_C, M, N, K);
  auto stop = std::chrono::high_resolution_clock::now();

  // Copy back 
  hipMemcpy(host_C, device_C, Mat_C_bytes, hipMemcpyDeviceToHost);

  // Check error
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::cout << "Error: " << error << std::endl;
  } else {
    print_matrix(host_A, M, N);
    std::cout << std::endl;
    print_matrix(host_B, N, K);
    std::cout << std::endl;
    print_matrix(host_C, M, K);
    std::cout << std::endl;
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Time spent:" << duration.count()  << "ms" << std::endl;
  }

  // Free memory 
  free(host_A);
  free(host_B);
  free(host_C);
  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);

  return 0;
}
   
