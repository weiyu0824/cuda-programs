#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <chrono>
#include <stdio.h>

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = threadIdx.x;
    if (i < N){
     C[i] = A[i] + B[i];
    }
}

int main(int argc, char** argv){

  hipDeviceProp_t deviceProp;
  printf("Device Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
  // Data size
  int N = 256;

  // Init 3 host memories
  float *host_A = (float*)malloc(N * sizeof(float));
  float *host_B = (float*)malloc(N * sizeof(float));
  float *host_C = (float*)malloc(N * sizeof(float));

  // Init 3 device memories
  float *device_A, *device_B, *device_C;
  hipMalloc((void**)&device_A, N * sizeof(float));
  hipMalloc((void**)&device_B, N * sizeof(float));
  hipMalloc((void**)&device_C, N * sizeof(float));

  // Init host data
  for (int i = 0; i < N; i ++) {
    host_A[i] = float(i);
    host_B[i] = float(i);
  }
  // std::cout << host_B[5];
  
  // Copy data from host -> device
  hipMemcpy(device_A, host_A, N * sizeof(float), hipMemcpyHostToDevice); 
  hipMemcpy(device_B, host_B, N * sizeof(float), hipMemcpyHostToDevice); 

  // Launch kernel
  auto start = std::chrono::high_resolution_clock::now();
  VecAdd<<<1, N>>>(device_A, device_B, device_C, N); 

  // Copy result from device -> host
  hipMemcpy(host_C, device_C, N * sizeof(float), hipMemcpyDeviceToHost); 
  auto stop = std::chrono::high_resolution_clock::now();  
  
  // Print result
  std::cout << "Result:";
  for (int i = 0; i < N; i ++) {
    std::cout << host_C[i] << " ";  
  }
  std::cout << std::endl;
  
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
  std::cout << "Time spent:" << duration.count()  << "ms" << std::endl;

  // Free memory
  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);
  free(host_A);
  free(host_B);
  free(host_C);

  return 0;
}
