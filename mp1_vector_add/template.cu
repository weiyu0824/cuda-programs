#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <chrono>
#include <stdio.h>

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N){
      C[i] = A[i] + B[i];
    }
}

int main(int argc, char** argv){

  // Data size
  int N = 10240;

  // Init 3 host memories
  float *host_A = (float*)malloc(N * sizeof(float));
  float *host_B = (float*)malloc(N * sizeof(float));
  float *host_C = (float*)malloc(N * sizeof(float));

  // Init 3 device memories
  float *device_A, *device_B, *device_C;
  hipMalloc((void**)&device_A, N * sizeof(float));
  hipMalloc((void**)&device_B, N * sizeof(float));
  hipMalloc((void**)&device_C, N * sizeof(float));

  // Init host data
  for (int i = 0; i < N; i ++) {
    host_A[i] = float(i);
    host_B[i] = float(i);
  }
  
  // Copy data from host -> device
  hipMemcpy(device_A, host_A, N * sizeof(float), hipMemcpyHostToDevice); 
  hipMemcpy(device_B, host_B, N * sizeof(float), hipMemcpyHostToDevice); 

  // 
  dim3 threadsPerBlock(32);
  dim3 numBlocks(N/threadsPerBlock.x);
  // Launch kernel
  auto start = std::chrono::high_resolution_clock::now();
  VecAdd<<<numBlocks, threadsPerBlock>>>(device_A, device_B, device_C, N); 

  // Copy result from device -> host
  hipMemcpy(host_C, device_C, N * sizeof(float), hipMemcpyDeviceToHost); 
  auto stop = std::chrono::high_resolution_clock::now();  
  
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
      printf("CUDA error: %s\n", hipGetErrorString(error));
      // Handle error
  }else{
    // Print result
    std::cout << "Result:";
    for (int i = 0; i < N; i ++) {
      std::cout << host_C[i] << " ";  
    }
    std::cout << std::endl;
    
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Time spent:" << duration.count()  << "ms" << std::endl;
  }
  // Free memory
  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);
  free(host_A);
  free(host_B);
  free(host_C);

  return 0;
}
