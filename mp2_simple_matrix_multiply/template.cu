
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <chrono>

__global__ void MatMul(float *A, float *B, float *C, int M, int N, int K){
  
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < M && j < K) {
    int val = 0;
    for (int seq_idx = 0; seq_idx < N; seq_idx ++){
      val += A[i * N + seq_idx] * B[j + seq_idx * N];
    }
    C[i * K + j] = val;
  }

} 

void print_matrix(float* matrix, int row, int col){
    for (int i = 0; i < row; i ++) {
      for (int j = 0; j < col; j ++) {
        int idx = i * col + j;
        std::cout << matrix[idx] << " "; 
      }
      std::cout << std::endl;
    } 
}

int main(){
  // Data size
  int M = 10;
  int N = 4;
  int K = 4; 
  
  size_t Mat_A_bytes = M * N * sizeof(float);
  size_t Mat_B_bytes = N * K * sizeof(float);
  size_t Mat_C_bytes = M * K * sizeof(float);

  // Init host memory
  float *host_A = (float*)malloc(Mat_A_bytes);
  float *host_B = (float*)malloc(Mat_B_bytes);
  float *host_C = (float*)malloc(Mat_C_bytes);
  
  // Init device memory
  float *device_A, *device_B, *device_C;
  hipMalloc((void**)&device_A, Mat_A_bytes);
  hipMalloc((void**)&device_B, Mat_B_bytes);
  hipMalloc((void**)&device_C, Mat_C_bytes);

  // Init Data
  for (int i = 0; i < M; i ++) {
    for (int j = 0; j < N; j ++) {
       int idx = i * N + j; 
       host_A[idx] = i;
    }
  } 
  for (int i = 0; i < N; i ++) {
    for (int j = 0; j < K; j ++) {
      int idx = i * K + j;
      host_B[idx] = i;
    }
  } 

  // Copy data to device
  hipMemcpy(device_A, host_A, Mat_A_bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_B, host_B, Mat_B_bytes, hipMemcpyHostToDevice);

  // Lanch kernel 
  dim3 blockDim(4, 4);
  int numBlockRows = (int)ceil(M / (float)blockDim.x);
  int numBlockCols = (int)ceil(K / (float)blockDim.y);
  dim3 gridDim(numBlockRows, numBlockCols);

  auto start = std::chrono::high_resolution_clock::now();
  MatMul<<<gridDim, blockDim>>>(device_A, device_B, device_C, M, N, K);
  auto stop = std::chrono::high_resolution_clock::now();
  
  // Copy back 
  hipMemcpy(host_C, device_C, Mat_C_bytes, hipMemcpyDeviceToHost);

  // Check error
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::cout << "Error: " << error << std::endl;
  } else {
    print_matrix(host_A, M, N);
    std::cout << std::endl;
    print_matrix(host_B, N, K);
    std::cout << std::endl;
    print_matrix(host_C, M, K);
    std::cout << std::endl;
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Time spent:" << duration.count()  << "ms" << std::endl;
  }

  // Free memory 
  free(host_A);
  free(host_B);
  free(host_C);
  hipFree(device_A);
  hipFree(device_B);
  hipFree(device_C);

  return 0;
}
